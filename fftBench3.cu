#include "hip/hip_runtime.h"
//  New benchmark for both a 1D and 2D forward and backward in-place complex FFT benchmark
//  using cuFFT and g++


// Includes, system
#include <iostream>
#include <cmath>

// Includes, project
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#define FFT_SIZE   65536
#define BATCH_SIZE 1

using namespace std;

//==========================================================================================

int main(int argc, char** argv)
{

    std::cout << "........ Starting cuFFT benchmark" << std::endl;
    
    //---- Step 1: Check for system GPU(s)
    //----------------------------------------------------------     
    int device_count = 0, device = -1;
    checkCudaErrors(hipGetDeviceCount(&device_count) );
    std::cout << "......... Found " << device_count << " devices" << std::endl;
    
    if (device_count > 0) {
        device = 0;
        checkCudaErrors(hipSetDevice(device));
    }
    else {
        std::cout << "Found error: No cuda devices present. Stopping." << std::endl;
        return(0);
    }
    
    
    //---- Step 2: Use only 1 GPU; set up cuda timers
    //----------------------------------------------------------
    hipEvent_t startF, stopF, startB, stopB;
    
    checkCudaErrors(hipEventCreate(&startF));
    checkCudaErrors(hipEventCreate(&stopF));     
    checkCudaErrors(hipEventCreate(&startB));
    checkCudaErrors(hipEventCreate(&stopB));   
    
    //---- Step 3: Set up the cufft plan; allocate host and device memory. 
    //----------------------------------------------------------   
    hipfftComplex *d_array = NULL;
    hipfftComplex *h_array = new hipComplex[FFT_SIZE];
    checkCudaErrors(hipMalloc((void**)&d_array, FFT_SIZE * sizeof(hipfftComplex)));
    
    hipfftHandle plan;
    checkCudaErrors( hipfftPlan1d(&plan, FFT_SIZE, HIPFFT_C2C, BATCH_SIZE) );
    
        
    //---- Step 4: Fill the allocated memory with values
    //----------------------------------------------------------
    for (int i = 0; i < FFT_SIZE; ++i) {
        h_array[i].x = 1.0;
        h_array[i].y = 1.0;
    }
       
    checkCudaErrors(hipMemcpy(d_array, h_array, FFT_SIZE * sizeof(hipfftComplex), hipMemcpyHostToDevice));
    
    //---- Step 5: Run the forward inplace FFT, check the values
    //----------------------------------------------------------    
    checkCudaErrors(hipEventRecord(startF, 0));
    
      checkCudaErrors(hipfftExecC2C(plan, d_array, d_array, HIPFFT_FORWARD));
        
    checkCudaErrors(hipEventRecord(stopF, 0));
    checkCudaErrors(hipDeviceSynchronize());
    
    // copy back to host; check answer
    checkCudaErrors(hipMemcpy(h_array, d_array, FFT_SIZE*sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventSynchronize(stopF));

    
    if (h_array[0].x != FFT_SIZE && h_array[0].y != FFT_SIZE ) {
        std::cout << " Error in forward FFT result: values returned are: (" << h_array[0].x << ", " << h_array[0].y << ")" << std::endl;
    }
    else {
        std::cout << "For now let's assume the forward FFT is ok...check to make certain everything else is 0.0 " << std::endl;
    }
    
    //---- Step 6: Run the backward inplace FFT, check the values (normalize?)
    //----------------------------------------------------------     
    checkCudaErrors(hipEventRecord(startB, 0));
    
      checkCudaErrors(hipfftExecC2C(plan, d_array, d_array, HIPFFT_BACKWARD));
        
    checkCudaErrors(hipEventRecord(stopB, 0));
    checkCudaErrors(hipDeviceSynchronize());
    
    // copy back to host; check answer
    checkCudaErrors(hipMemcpy(h_array, d_array, FFT_SIZE*sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventSynchronize(stopB)); 
    
    int numErrors = 0;
    for (int i = 0; i< FFT_SIZE; ++i){ 
        if (h_array[i].x != FFT_SIZE) numErrors+=1;  // Remember...we didn't normalize.
        if (h_array[i].y != FFT_SIZE) numErrors+=1;   
    }
    if (numErrors > 0)
        std::cout << " !!!! Found " << numErrors << " errors in Backward FFT !!!!! " << std::endl;
        
/*        
    for (int i = 0; i < 10; ++i) {
       std::cout << "( " << (float)h_array[i].x << " , " << (float)h_array[i].y << " )" << std::endl;
    }
*/
    
    //---- Step 7: Timing results
    //----------------------------------------------------------    
    float millisecsF = 0.;
    float millisecsB = 0.;
    hipEventElapsedTime(&millisecsF, startF, stopF);
    hipEventElapsedTime(&millisecsB, startB, stopB);
    
    
    std::cout << "...... Forward FFT:   " << millisecsF * 1.e3 << " microsecs" << std::endl;
    std::cout << "...... Backward FFT:  " << millisecsB * 1.e3 << " microsecs" << std::endl;
    
    int nFLOPS = 5 * FFT_SIZE * BATCH_SIZE * log2( FFT_SIZE * BATCH_SIZE);
    
    std::cout << ".....  Forward computational throughput:  " << (float)nFLOPS/(millisecsF * 1.e-3) * 1.e-9 << " GFlops" << std::endl;
    std::cout << ".....  Inverse computational throughput:  " << (float)nFLOPS/(millisecsB * 1.e-3) * 1.e-9 << " GFlops" << std::endl;
    
    //---- Step 8: Cleanup allocated memory
    //---------------------------------------------------------- 
    delete[]  h_array;
    checkCudaErrors(hipFree(d_array));
    checkCudaErrors(hipfftDestroy(plan));
    
    
    
    std::cout << "........ Finishing cuFFT benchmark" << std::endl;
    return( 0 );
}

